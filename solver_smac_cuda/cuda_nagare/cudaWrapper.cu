#include "cuda_nagare/cudaWrapper.cuh"

#include "variables.hpp"

#include <iostream>
#include <vector>
#include <list>

namespace cudaWrapper {
    // Malloc
    void cudaMalloc_wrapper(flow_float** var_d , geom_int size)
    {
        CHECK_CUDA_ERROR( hipMalloc(var_d , size*sizeof(flow_float)) );
    };

    void cudaMalloc_wrapper(geom_int** var_d , geom_int size)
    {
        CHECK_CUDA_ERROR( hipMalloc(var_d , size*sizeof(geom_int)) );
    };

    // Memcpy
    //void cudaMemcpy_vectorToDevice_wrapper(std::vector<flow_float>& vec , flow_float* var_d)
    //{
    //    CHECK_CUDA_ERROR( hipMemcpy(var_d, vec.data() , vec.size()*sizeof(flow_float), hipMemcpyHostToDevice) );
    //};

    //void cudaMemcpy_deviceToVector_wrapper(flow_float* var_d , std::vector<flow_float>& vec )
    //{
    //    CHECK_CUDA_ERROR( hipMemcpy(vec.data() , var_d, vec.size()*sizeof(flow_float), hipMemcpyDeviceToHost) );
    //};

    void cudaMemcpy_H2D_wrapper(flow_float* vec , flow_float* var_d , geom_int numEle)
    {
        CHECK_CUDA_ERROR( hipMemcpy(var_d, vec , (size_t)(numEle*sizeof(flow_float)), hipMemcpyHostToDevice) );
    };

    void cudaMemcpy_D2H_wrapper(flow_float* var_d , flow_float* vec , geom_int numEle)
    {
        CHECK_CUDA_ERROR( hipMemcpy(vec, var_d, numEle*sizeof(flow_float), hipMemcpyDeviceToHost) );
    };


    // free
    void cudaFree_wrapper(flow_float* var_d)
    {
        CHECK_CUDA_ERROR( hipFree(var_d) );
    };

    void cudaFree_wrapper(geom_int* var_d)
    {
        CHECK_CUDA_ERROR( hipFree(var_d) );
    };

    //void copyVariables_cell_plane_H2D(variables& var)
    //{
    //    for (auto& name : var.cellValNames)
    //    {
    //        cudaMemcpy_vectorToDevice_wrapper(var.c[name] , var.c_d[name]);
    //    }
    //    for (auto& name : var.planeValNames)
    //    {
    //        cudaMemcpy_vectorToDevice_wrapper(var.p[name] , var.p_d[name]);
    //    }
    //}
};
